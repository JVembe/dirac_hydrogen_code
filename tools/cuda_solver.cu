#include "hip/hip_runtime.h"
#include "cuda_translate.h"

#include <unistd.h>

extern "C" {
#include "csr.h"
#include "gpu_sparse.h"
#include "solver_common.h"
#include "../src/tictoc.h"
}

extern int rank;
extern size_t total_bytes;

extern "C" {

    static int submatrix_nnz;
    static __device__ gpu_sparse_csr_t Hfull;
    static __device__ gpu_sparse_csr_t *g = NULL;
    static __device__ gpu_sparse_csr_t *gt = NULL;
    static __device__ gpu_sparse_csr_t *h0 = NULL;
    static __device__ csr_data_t *pgsum = NULL;
    static __device__ csr_data_t *pgtsum = NULL;
    static __device__ csr_index_t *Ai_orig, *Ki;
    static __device__ csr_index_t *Aj_orig, *Kj;
    static __device__ csr_data_t *H0, *H1;

    __global__ void device_compute_pgsums(csr_data_t cihdt , int lmax, int subsize,
                                          csr_data_t ft0, csr_data_t ft1, csr_data_t ft2,
                                          csr_data_t ft3, csr_data_t ft4, csr_data_t ft5)
    {
        const uint ix = blockIdx.x*blockDim.x + threadIdx.x;
        const uint gx = blockIdx.y;
        const uint lx = blockIdx.z;
        csr_data_t ft[6] = {ft0, ft1, ft2, ft3, ft4, ft5};
        csr_data_t pgtmp, pgttmp;
        const csr_data_t *pg, *pgt;
        uint ax;

        ax = (lx%2+1)%2;
        pg = g[ax*4*lmax + lx*4 + gx].Ax;
        pgtmp = hipCmul(pg[ix], hipCmul(cihdt, ft[ax]));
        pgt = gt[ax*4*lmax + lx*4 + gx].Ax;
        pgttmp = hipCmul(pgt[ix], hipCmul(cihdt, ft[ax]));

        ax += 2;
        pg = g[ax*4*lmax + lx*4 + gx].Ax;
        pgtmp = hipCadd(pgtmp, hipCmul(pg[ix], hipCmul(cihdt, ft[ax])));
        pgt = gt[ax*4*lmax + lx*4 + gx].Ax;
        pgttmp = hipCadd(pgttmp, hipCmul(pgt[ix], hipCmul(cihdt, ft[ax])));

        ax += 2;
        pg = g[ax*4*lmax + lx*4 + gx].Ax;
        pgtmp = hipCadd(pgtmp, hipCmul(pg[ix], hipCmul(cihdt, ft[ax])));
        pgt = gt[ax*4*lmax + lx*4 + gx].Ax;
        pgttmp = hipCadd(pgttmp, hipCmul(pgt[ix], hipCmul(cihdt, ft[ax])));

        pgsum[lx*4*subsize + gx*subsize + ix] = pgtmp;
        pgtsum[lx*4*subsize + gx*subsize + ix] = pgttmp;
    }

    __global__ void device_compute_timedep_matrices(int lmax, hipDoubleComplex ihdt, csr_index_t submatrix_nnz, csr_index_t subsize)
    {
        const uint thrid  = threadIdx.x;
        const uint nthr   = blockDim.x;
        const uint nnzid  = blockIdx.x; // non-zero entry in the Hfull_blk matrix
        const uint subblk = blockIdx.y; // part of the submatrix this thread works on
        const uint submatrix_loc = subblk*nthr + thrid;  // id of the submatrix non-zero entry this thread computes
        const uint map_loc = nnzid*submatrix_nnz + submatrix_loc; // id of the global non-zero entry this thread computes

        // multipliers
        const hipDoubleComplex ki = make_hipDoubleComplex(Ki[nnzid], 0);
        const hipDoubleComplex kj = make_hipDoubleComplex(Kj[nnzid], 0);
        const hipDoubleComplex ki2 = hipCmul(ki, ki);
        const hipDoubleComplex ki3 = hipCmul(ki2, ki);
        const hipDoubleComplex kikj = hipCmul(ki, kj);

        hipDoubleComplex result, tmp;

        // deactivate threads outside of submatrix range
        if(submatrix_loc < submatrix_nnz){

            // diagonal part
            result = make_hipDoubleComplex(0.0, 0.0);
            if(Ai_orig[nnzid] == Aj_orig[nnzid]){
                tmp = hipCadd(hipCmul(h0[1].Ax[submatrix_loc], ki), h0[0].Ax[submatrix_loc]);
                tmp = hipCadd(hipCmul(h0[2].Ax[submatrix_loc], ki2), tmp);
                tmp = hipCadd(hipCmul(h0[3].Ax[submatrix_loc], ki3), tmp);
                result = hipCmul(ihdt, tmp);
            }

            for(int l=0; l<lmax; l++){
                hipDoubleComplex H0l = H0[nnzid*lmax + l];
                tmp = pgsum[l*4*subsize + 0*subsize + submatrix_loc];
                tmp = hipCadd(tmp, hipCmul(pgsum[l*4*subsize + 1*subsize + submatrix_loc], ki));
                tmp = hipCadd(tmp, hipCmul(pgsum[l*4*subsize + 2*subsize + submatrix_loc], kj));
                tmp = hipCadd(tmp, hipCmul(pgsum[l*4*subsize + 3*subsize + submatrix_loc], kikj));
                tmp = hipCmul(tmp, H0l);
                result = hipCadd(result, tmp);

                hipDoubleComplex H1l = H1[nnzid*lmax + l];
                tmp = pgtsum[l*4*subsize + 0*subsize + submatrix_loc];
                tmp = hipCadd(tmp, hipCmul(pgtsum[l*4*subsize + 1*subsize + submatrix_loc], kj));
                tmp = hipCadd(tmp, hipCmul(pgtsum[l*4*subsize + 2*subsize + submatrix_loc], ki));
                tmp = hipCadd(tmp, hipCmul(pgtsum[l*4*subsize + 3*subsize + submatrix_loc], kikj));
                tmp = hipCmul(tmp, H1l);
                result = hipCadd(result, tmp);
            }

            uint dest = Hfull.Ai_sub_map[map_loc];
            Hfull.Ax[dest] = result;
        }
    }

    void cuda_compute_row_col(int lmax, const sparse_csr_t *H,
                              sparse_csr_t *Hfull_blk, sparse_csr_t *Hfull)
    {
        csr_index_t nrows = csr_nrowblocks(Hfull_blk);
        csr_index_t row, col, colp;

        csr_index_t *cpu_Ai_orig, *cpu_Ki;
        csr_index_t *cpu_Aj_orig, *cpu_Kj;
        csr_data_t *cpu_H0, *cpu_H1;
        csr_index_t Hfull_nnz = Hfull_blk->nnz;

        // there arrays have O(Hfull_blk.nnz) entries
        cpu_Ai_orig = (csr_index_t*)malloc(sizeof(csr_index_t)*Hfull_nnz);
        cpu_Aj_orig = (csr_index_t*)malloc(sizeof(csr_index_t)*Hfull_nnz);
        cpu_Ki = (csr_index_t*)malloc(sizeof(csr_index_t)*Hfull_nnz);
        cpu_Kj = (csr_index_t*)malloc(sizeof(csr_index_t)*Hfull_nnz);
        cpu_H0 = (csr_data_t*)malloc(sizeof(csr_data_t)*Hfull_nnz*lmax);
        cpu_H1 = (csr_data_t*)malloc(sizeof(csr_data_t)*Hfull_nnz*lmax);

        for(row = 0; row < nrows; row++){

            // for non-zeros in each row
            for(colp = Hfull_blk->Ap[row]; colp < Hfull_blk->Ap[row+1]; colp++){

                // NOTE: rows and cols in Hfull_blk are remapped wrt. the original numbering in H
                col = Hfull_blk->Ai[colp];

                // apply node renumbering - if available
                csr_index_t orig_row = row;
                csr_index_t orig_col = col;
                if(Hfull_blk->perm) {
                    orig_row = Hfull_blk->perm[csr_local_rowoffset(Hfull_blk) + row];
                    orig_col = Hfull_blk->perm[col];
                }

                // calculate kappa and mu parameters from row/col indices
                // see spnrbasis::bdpalphsigmaXmat
                int ki = (int)ik(orig_row); // k'
                int kj = (int)ik(orig_col); // k

                cpu_Ai_orig[colp] = orig_row;
                cpu_Aj_orig[colp] = orig_col;
                cpu_Ki[colp] = ki;
                cpu_Kj[colp] = kj;

                // prefetch the Hamiltonian values H0(l) and H1(l)
                for(int l=0; l<lmax; l++){
                    cpu_H0[colp*lmax + l] = csr_get_value(H + 2*l + 0, orig_row, orig_col);
                    cpu_H1[colp*lmax + l] = csr_get_value(H + 2*l + 1, orig_row, orig_col);
                }
            }
        }

        // copy the auxiliary arrays to device
        {
            void *devp;
            size_t bytes = 0;

            // Ai_orig
            CHECK_GPU(hipMalloc((void**) &devp, sizeof(csr_index_t)*Hfull_nnz));
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(Ai_orig), &devp, sizeof(csr_index_t*), 0, hipMemcpyHostToDevice));
            CHECK_GPU(hipMemcpy(devp, cpu_Ai_orig, sizeof(csr_index_t)*Hfull_nnz, hipMemcpyHostToDevice));
            bytes += sizeof(csr_index_t)*Hfull_nnz;

            // Aj_orig
            CHECK_GPU(hipMalloc((void**) &devp, sizeof(csr_index_t)*Hfull_nnz));
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(Aj_orig), &devp, sizeof(csr_index_t*), 0, hipMemcpyHostToDevice));
            CHECK_GPU(hipMemcpy(devp, cpu_Aj_orig, sizeof(csr_index_t)*Hfull_nnz, hipMemcpyHostToDevice));
            bytes += sizeof(csr_index_t)*Hfull_nnz;

            // Ki
            CHECK_GPU(hipMalloc((void**) &devp, sizeof(csr_index_t)*Hfull_nnz));
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(Ki), &devp, sizeof(csr_index_t*), 0, hipMemcpyHostToDevice));
            CHECK_GPU(hipMemcpy(devp, cpu_Ki, sizeof(csr_index_t)*Hfull_nnz, hipMemcpyHostToDevice));
            bytes += sizeof(csr_index_t)*Hfull_nnz;

            // Kj
            CHECK_GPU(hipMalloc((void**) &devp, sizeof(csr_index_t)*Hfull_nnz));
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(Kj), &devp, sizeof(csr_index_t*), 0, hipMemcpyHostToDevice));
            CHECK_GPU(hipMemcpy(devp, cpu_Kj, sizeof(csr_index_t)*Hfull_nnz, hipMemcpyHostToDevice));
            bytes += sizeof(csr_index_t)*Hfull_nnz;

            // H0
            CHECK_GPU(hipMalloc((void**) &devp, sizeof(csr_data_t)*Hfull_nnz*lmax));
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(H0), &devp, sizeof(csr_data_t*), 0, hipMemcpyHostToDevice));
            CHECK_GPU(hipMemcpy(devp, cpu_H0, sizeof(csr_data_t)*Hfull_nnz*lmax, hipMemcpyHostToDevice));
            bytes += sizeof(csr_data_t)*Hfull_nnz*lmax;

            // H1
            CHECK_GPU(hipMalloc((void**) &devp, sizeof(csr_data_t)*Hfull_nnz*lmax));
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(H1), &devp, sizeof(csr_data_t*), 0, hipMemcpyHostToDevice));
            CHECK_GPU(hipMemcpy(devp, cpu_H1, sizeof(csr_data_t)*Hfull_nnz*lmax, hipMemcpyHostToDevice));
            bytes += sizeof(csr_data_t)*Hfull_nnz*lmax;

            total_bytes += bytes;
            PDEBUG("cuda_compute_row_col: allocated %li bytes on GPU (total %li)\n", bytes, total_bytes);

            printf("CUDA copied auxiliary arrays to GPU\n");
        }
    }

    static int is_timedep_initialized = 0;
    void cuda_compute_timedep_matrices(double h, double dt, csr_data_t *ft, int lmax,
                                       const sparse_csr_t *cpu_Hfull_blk, sparse_csr_t *cpu_Hfull,
                                       gpu_sparse_csr_t *gpu_Hfull)
    {
        int nthreads = 64;
        int nblocks = submatrix_nnz/nthreads;
        if(nblocks*nthreads<submatrix_nnz) nblocks++;
        int subsize = nblocks*nthreads;

        // one-time init - memory allocation
        if(!is_timedep_initialized){

            size_t bytes = 0;
            
            // copy the matrix structure
            gpu_put_csr(gpu_Hfull, cpu_Hfull);
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(Hfull), gpu_Hfull, sizeof(gpu_sparse_csr_t), 0, hipMemcpyHostToDevice));

            csr_data_t *devp;
            CHECK_GPU(hipMalloc((void**) &devp, lmax*4*subsize*sizeof(csr_data_t)));
            bytes += lmax*4*subsize*sizeof(csr_data_t);
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(pgsum), &devp, sizeof(csr_data_t*), 0, hipMemcpyHostToDevice));

            CHECK_GPU(hipMalloc((void**) &devp, lmax*4*subsize*sizeof(csr_data_t)));
            bytes += lmax*4*subsize*sizeof(csr_data_t);
            CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(pgtsum), &devp, sizeof(csr_data_t*), 0, hipMemcpyHostToDevice));

            is_timedep_initialized = 1;

            total_bytes += bytes;
            PDEBUG("cuda_compute_timedep_matrices: allocated %li bytes on GPU (total %li)\n", bytes, total_bytes);
        }

        // precompute G-sums and Gt-sums
        {
            dim3 blocks(nblocks, 4, lmax);
            csr_data_t cihdt = make_hipDoubleComplex(0.0, SoL*h*dt/2);
            device_compute_pgsums<<<blocks, nthreads>>>(cihdt, lmax, subsize, ft[0], ft[1], ft[2], ft[3], ft[4], ft[5]);
        }

        // assemble Ht
        {
            dim3 blocks(cpu_Hfull_blk->nnz, nblocks);
            csr_data_t ihdt = make_hipDoubleComplex(0.0, h*dt/2);
            device_compute_timedep_matrices<<<blocks, nthreads>>>(lmax, ihdt, submatrix_nnz, subsize);
        }
        hipDeviceSynchronize();
    }

    void cuda_init_model_matrices(int nmatrices, const sparse_csr_t *cpu_g, const sparse_csr_t *cpu_gt, const sparse_csr_t *cpu_h0)
    {
        gpu_sparse_csr_t gputemp = {};
        gpu_sparse_csr_t *devp;
        size_t bytes = 0;

        submatrix_nnz = cpu_g[0].nnz;

        CHECK_GPU(hipMalloc((void**) &devp, nmatrices*sizeof(gpu_sparse_csr_t)));
        bytes += nmatrices*sizeof(gpu_sparse_csr_t);
        CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(g), &devp, sizeof(gpu_sparse_csr_t*), 0, hipMemcpyHostToDevice));
        for(int i=0; i<nmatrices; i++){
            memset(&gputemp, 0, sizeof(gpu_sparse_csr_t));
            gpu_put_csr(&gputemp, cpu_g+i);
            CHECK_GPU(hipMemcpy(devp+i, &gputemp, sizeof(gpu_sparse_csr_t), hipMemcpyHostToDevice));
        }

        CHECK_GPU(hipMalloc((void**) &devp, nmatrices*sizeof(gpu_sparse_csr_t)));
        bytes += nmatrices*sizeof(gpu_sparse_csr_t);
        CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(gt), &devp, sizeof(gpu_sparse_csr_t*), 0, hipMemcpyHostToDevice));
        for(int i=0; i<nmatrices; i++){
            memset(&gputemp, 0, sizeof(gpu_sparse_csr_t));
            gpu_put_csr(&gputemp, cpu_gt+i);
            CHECK_GPU(hipMemcpy(devp+i, &gputemp, sizeof(gpu_sparse_csr_t), hipMemcpyHostToDevice));
        }

        CHECK_GPU(hipMalloc((void**) &devp, 4*sizeof(gpu_sparse_csr_t)));
        bytes += nmatrices*sizeof(gpu_sparse_csr_t);
        CHECK_GPU(hipMemcpyToSymbol(HIP_SYMBOL(h0), &devp, sizeof(gpu_sparse_csr_t*), 0, hipMemcpyHostToDevice));
        for(int i=0; i<4; i++){
            memset(&gputemp, 0, sizeof(gpu_sparse_csr_t));
            gpu_put_csr(&gputemp, cpu_h0+i);
            CHECK_GPU(hipMemcpy(devp+i, &gputemp, sizeof(gpu_sparse_csr_t), hipMemcpyHostToDevice));
        }

        total_bytes += bytes;
        PDEBUG("cuda_init_model_matrices: allocated %li bytes on GPU (total %li)\n", bytes, total_bytes);
        
        printf("CUDA model matrices initialized\n");
    }
}
